#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
 
#include "config.h"

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}
                
// Add GPU kernel and functions
// HERE!!!
__global__ void kernel(unsigned char *input, 
                       unsigned char *output,
                       int           *lookup,
                       unsigned int width,
                       unsigned int height){

    int x = blockIdx.x*TILE_SIZE+threadIdx.x;
    int y = blockIdx.y*TILE_SIZE+threadIdx.y;
    int location = 	y*TILE_SIZE*gridDim.x+x;


	int size = width*height;
	float_t alpha = 255/size;
	
    // calculate histogram 
    if (location < size){
		lookup[input[location]] +=1;
	}
	
    __syncthreads();

    
    // calcualte cumultaive histogram nomralized to 0~255
    lookup[0] = lookup[0]*alpha;
    __syncthreads();
    
	for(int i=1; i<256; i++) {
		lookup[i] += lookup[i-1]*alpha;
		
	}
    __syncthreads();
    
    
    // calculate 
    if (location < size){
		output[location] = lookup[input[location]];
	}
	

}

__global__ void warmup(unsigned char *input, 
                       unsigned char *output){

	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
	int location = 	y*(gridDim.x*TILE_SIZE)+x;
	
    output[location] = 0;

}

// NOTE: The data passed on is already padded
void gpu_function(unsigned char *data,  
                  unsigned int height, 
                  unsigned int width){
    
    unsigned char *input_gpu;
    unsigned char *output_gpu;
	int *lookup;// might pass in the lookup table preinitalized to zero instead

	

	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
    checkCuda(hipMalloc((void**)&lookup  , 256*sizeof(int)));
    
    checkCuda(hipMemset(lookup , 0 , 256*sizeof(int)));
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
	
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu, 
        data, 
        size*sizeof(char), 
        hipMemcpyHostToDevice));

        
	checkCuda(hipDeviceSynchronize());

    // Execute algorithm

    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

	// Kernel Call
	#ifdef CUDA_TIMING
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
        
        printf("here:/n");
        // Add more kernels and functions as needed here
        kernel<<<dimGrid, dimBlock>>>(input_gpu, 
                                      output_gpu,
                                      lookup,
                                      width,
                                      height);
        
        // From here on, no need to change anything
        checkCuda(hipPeekAtLastError());                                     
        checkCuda(hipDeviceSynchronize());
	
	#ifdef CUDA_TIMING
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));

    // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));

}

void gpu_warmup(unsigned char *data, 
                unsigned int height, 
                unsigned int width){
    
    unsigned char *input_gpu;
    unsigned char *output_gpu;
     
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	// Both are the same size (CPU/GPU).
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
	
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
            
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu, 
        data, 
        size*sizeof(char), 
        hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());
        
    // Execute algorithm
        
	dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    
    warmup<<<dimGrid, dimBlock>>>(input_gpu, 
                                  output_gpu);
                                         
    checkCuda(hipDeviceSynchronize());
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));
                        
    // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));

}

